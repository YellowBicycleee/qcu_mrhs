#include <hip/hip_fp16.h>

#include "kernel/su_n_m_rhs_dslash.cuh"
#include "qcd/qcu_dslash.h"
#include "qcu_macro.h"
#include "qcu_wmma_constant.h"
namespace qcu {

// clang-format off
template <typename Float>
inline void ApplyWilsonDslash_Mrhs( Float* __restrict__ out, Float* __restrict__ in, Float* __restrict__ gauge, 
                                    int Lx, int Ly, int Lz, int Lt, int g_x, int g_y, int g_z, int g_t, 
                                    int parity, bool dagger_flag, int n_color, int m_rhs, hipStream_t& stream
) {
    // clang-format on
    // constexpr int WMMA_M = device::WMMA_Param<Float>::WMMA_M;
    // constexpr int WMMA_N = device::WMMA_Param<Float>::WMMA_N;
    // constexpr int WMMA_K = device::WMMA_Param<Float>::WMMA_K;

    // clang-format off
    // constexpr int smem_size_per_warp = 2 * (
    //                                    WMMA_M * WMMA_K  +         /* U                          wmma_m * wmma_k */
    //                                    WMMA_K * WMMA_N  +         /* T1                         wmma_k * wmma_n */
    //                               6 * (WMMA_M * WMMA_N)          /* R1, R2, L1, L2, L3, L4     wmma_m * wmma_n */
    //                               );
    // clang-format on

    int vol = Lx * Ly * Lz * Lt / 2;
    int warp_num_per_block = WARP_PER_BLOCK;
    // size_t smem_size = warp_num_per_block * smem_size_per_warp * sizeof(Float);

    dim3 block_size(WARP_SIZE, warp_num_per_block);
    dim3 grid_size(vol);
    device::wilson_dslash_su_n_mrhs<Float> <<<grid_size, block_size, 0, stream>>>(
        out, in, gauge, Lx, Ly, Lz, Lt, g_x, g_y, g_z, g_t, parity, dagger_flag, n_color, m_rhs);
    
    // wilson_dslash_su_n_mrhs(Float* __restrict__ out, Float* __restrict__ in, Float* __restrict__ gauge,
    //                                     int Lx, int Ly, int Lz, int Lt, int g_x, int g_y, int g_z, int g_t, int parity,
    //                                     bool dagger_flag, int n_color, int m_rhs) 
}

void WilsonDslash::apply() {
    int Lx = dslashParam_.lattDesc.dims[X_DIM];
    int Ly = dslashParam_.lattDesc.dims[Y_DIM];
    int Lz = dslashParam_.lattDesc.dims[Z_DIM];
    int Lt = dslashParam_.lattDesc.dims[T_DIM];

    int g_x = dslashParam_.procDesc.dims[X_DIM];
    int g_y = dslashParam_.procDesc.dims[Y_DIM];
    int g_z = dslashParam_.procDesc.dims[Z_DIM];
    int g_t = dslashParam_.procDesc.dims[T_DIM];

    // clang-format off
    switch (dslashParam_.precision) {
        case QCU_HALF_PRECISION:
            ApplyWilsonDslash_Mrhs<half>(static_cast<half*>(dslashParam_.fermionOut_MRHS), 
                                         static_cast<half*>(dslashParam_.fermionIn_MRHS), 
                                         static_cast<half*>(dslashParam_.gauge), 
                                         Lx, Ly, Lz, Lt, g_x, g_y, g_z, g_t, 
                                         dslashParam_.parity, dslashParam_.daggerFlag, 
                                         dslashParam_.nColor, dslashParam_.mInput, 
                                         dslashParam_.stream1);
            /* code */
            break;
        case QCU_SINGLE_PRECISION:
            errorQcu("Not implemented yet\n");  // TODO
            assert(0);
            break;
        case QCU_DOUBLE_PRECISION:
            ApplyWilsonDslash_Mrhs<double>(static_cast<double*>(dslashParam_.fermionOut_MRHS), 
                                           static_cast<double*>(dslashParam_.fermionIn_MRHS), 
                                           static_cast<double*>(dslashParam_.gauge), 
                                           Lx, Ly, Lz, Lt, g_x, g_y, g_z, g_t, 
                                           dslashParam_.parity, dslashParam_.daggerFlag, 
                                           dslashParam_.nColor, dslashParam_.mInput, 
                                           dslashParam_.stream1); 
            break;
        default:
            errorQcu("Not implemented yet\n");  // TODO
            assert(0);
            break;
    }
    // clang-format on
}
void WilsonDslash::preApply() {
    errorQcu("Not implemented yet\n");  // TODO
}
void WilsonDslash::postApply() {
    errorQcu("Not implemented yet\n");  // TODO
}
// TODO : calc flops
void WilsonDslash::flops() {
    errorQcu("Not implemented yet\n");  // TODO
}

}  // namespace qcu