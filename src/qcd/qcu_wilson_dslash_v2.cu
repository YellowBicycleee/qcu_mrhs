#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>

#include "check_error/check_cuda.cuh"
#include "kernel/su_n_m_rhs_dslash_new.cuh"
#include "qcd/qcu_dslash_wilson.h"
#include "qcu_base/qcu_alloc.h"
#include "qcu_config/qcu_config.h"
#include "qcu_public.h"

#include "qcu_base/qcu_base.h"
/** how to impl dslash
 * Policy1: MPI Naive
 *    main thread-flag=false----------stream9--------------------------------------internal Kernel----flag=true--------join, sync---flag=false
 *                          |----sub_thread1(stream1)---EXTERNAL KERNEL---D2H---MPI---H2D----MPI_Test--|---->stream9------|
 *                          |----sub_thread2(stream2)---EXTERNAL KERNEL---D2H---MPI---H2D----MPI_Test--|---->stream9------|
 *                          |----sub_thread3(stream3)---EXTERNAL KERNEL---D2H---MPI---H2D----MPI_Test--|---->stream9------|
 *                          |----sub_thread4(stream4)---EXTERNAL KERNEL---D2H---MPI---H2D----MPI_Test--|---->stream9------|
 *                          |----sub_thread5(stream5)---EXTERNAL KERNEL---D2H---MPI---H2D----MPI_Test--|---->stream9------|
 *                          |----sub_thread6(stream6)---EXTERNAL KERNEL---D2H---MPI---H2D----MPI_Test--|---->stream9------|
 *                          |----sub_thread7(stream7)---EXTERNAL KERNEL---D2H---MPI---H2D----MPI_Test--|---->stream9------|
 *                          |----sub_thread8(stream8)---EXTERNAL KERNEL---D2H---MPI---H2D----MPI_Test--|---->stream9------|
 *                          |---------------------------------------------------------------------------------------------|
 * Policy2: MPI + NCCL
 */

namespace qcu::developing {
template <typename Float>
inline void ApplyWilsonDslash_Mrhs( DslashParam& dslash_param)
{
    int half_vol = config::lattice_volume() / 2;

    const qcu::QcuLattDesc& latt_desc = *(dslash_param.lattDesc);
    const qcu::QcuProcDesc& proc_desc = *(dslash_param.procDesc);

    using BlockShape = gemm::GemmShape<8, 8, 8>;
    // using BlockShape = gemm::GemmShape<8, 4, 4>;
    int multiprocess = 0;
    for (int i = 0; i < Nd; ++i) {
        if (proc_desc.data[i] > 0) {
            multiprocess |= (1 << i);
        }
    }

    int blk_x = BlockShape::kM;
    int blk_y = BlockShape::kN;

    dim3 grid_size(1, 1, min(half_vol, 65535));
    dim3 block_size(blk_x, blk_y, 1);

    printf("SIMT dslash Beginning\n");
    qcu::device::wilson_dslash_su_n_mrhs<Float, BlockShape>
        <<<grid_size, block_size, 0, dslash_param.stream1>>>
        (   static_cast<Float*>(dslash_param.fermionOut_MRHS),
            static_cast<Float*>(dslash_param.fermionIn_MRHS),
            static_cast<Float*>(dslash_param.gauge),
            latt_desc, multiprocess,
            dslash_param.parity, dslash_param.daggerFlag,
            dslash_param.nColor, dslash_param.mInput);
    CHECK_CUDA(hipDeviceSynchronize());
}

void WilsonDslash::apply(std::shared_ptr<DslashParam> dslash_param) {

    // clang-format off
    switch (dslash_param->precision) {
        case QcuPrecision::kPrecisionHalf:
            { ApplyWilsonDslash_Mrhs<half>(*dslash_param); }
            break;
        case QcuPrecision::kPrecisionSingle:
            {
                errorQcu("Not implemented yet\n");  // TODO
                assert(0);
            }
            break;
        case QcuPrecision::kPrecisionDouble:
            { ApplyWilsonDslash_Mrhs<double>(*dslash_param);}
            break;
        default:
            {
                errorQcu("Not implemented yet\n");  // TODO
                assert(0);
            }
            break;
    }
    CHECK_CUDA(hipStreamSynchronize(dslash_param->stream1));
}
void WilsonDslash::pre_apply(const std::shared_ptr<DslashParam> dslash_param) {
    errorQcu("Not implemented yet\n");  // TODO
}
void WilsonDslash::post_apply(const std::shared_ptr<DslashParam> dslash_param) {
    errorQcu("Not implemented yet\n");  // TODO
}
// TODO : calc flops
double WilsonDslash::flops() {
    errorQcu("Not implemented yet\n");  // TODO
    if (if_metric_) {
        return operations_ / time_;
    } else {
        return 0;
    }
}

}