#include <hip/hip_fp16.h>

#include <cassert>
#include <cstdlib>

#include "data_format/qcu_data_format_shift.cuh"
#include "qcu_interface.h"
#include "qcu_macro.h"
#include "timer/timer.h"
#include "qcu_wmma_constant.h"  // use this to debug
#include "qcu_utils.h"  // div_ceil
namespace qcu {

void Qcu::allocateMemory() {
    int Lx = lattDesc_.dims[X_DIM];
    int Ly = lattDesc_.dims[Y_DIM];
    int Lz = lattDesc_.dims[Z_DIM];
    int Lt = lattDesc_.dims[T_DIM];

    int vol = Lx * Ly * Lz * Lt;
    int colorSpinorMrhs_size = vol * Ns * nColors_ * mInput_;  // even and odd
    int gauge_size = Nd * vol * nColors_ * nColors_;   // even and odd

    switch (dslashFloatPrecision_) {
        case QCU_HALF_PRECISION: {
            CHECK_CUDA(hipMalloc(&fermionIn_MRHS_, 2 * colorSpinorMrhs_size * sizeof(half)));
            CHECK_CUDA(hipMalloc(&fermionOut_MRHS_, 2 * colorSpinorMrhs_size * sizeof(half)));
        } break;
        case QCU_SINGLE_PRECISION: {
            CHECK_CUDA(hipMalloc(&fermionIn_MRHS_, 2 * colorSpinorMrhs_size * sizeof(float)));
            CHECK_CUDA(hipMalloc(&fermionOut_MRHS_, 2 * colorSpinorMrhs_size * sizeof(float)));
        } break;
        case QCU_DOUBLE_PRECISION: {
            CHECK_CUDA(hipMalloc(&fermionIn_MRHS_, 2 * colorSpinorMrhs_size * sizeof(double)));
            CHECK_CUDA(hipMalloc(&fermionOut_MRHS_, 2 * colorSpinorMrhs_size * sizeof(double)));
        } break;

        default:
            break;
    }
    // gauge field
    CHECK_CUDA(hipMalloc(&fp64Gauge_, 2 * gauge_size * sizeof(double)));
    CHECK_CUDA(hipMalloc(&fp32Gauge_, 2 * gauge_size * sizeof(float)));
    CHECK_CUDA(hipMalloc(&fp16Gauge_, 2 * gauge_size * sizeof(half)));
}

void Qcu::freeMemory() {
    if (dslashParam_ != nullptr) {
        delete dslashParam_;
    }
    if (dslash_ != nullptr) {
        delete dslash_;
    }

    if (fp64Gauge_ != nullptr) {
        CHECK_CUDA(hipFree(fp64Gauge_));
    }
    if (fp32Gauge_ != nullptr) {
        CHECK_CUDA(hipFree(fp32Gauge_));
    }
    if (fp16Gauge_ != nullptr) {
        CHECK_CUDA(hipFree(fp16Gauge_));
    }
    if (fermionIn_MRHS_ != nullptr) {
        CHECK_CUDA(hipFree(fermionIn_MRHS_));
    }
    if (fermionOut_MRHS_ != nullptr) {
        CHECK_CUDA(hipFree(fermionOut_MRHS_));
    }
}

void Qcu::getDslash(DSLASH_TYPE dslashType, double mass) {
    if (nullptr != dslash_) {
        delete dslash_;
    }
    void* gauge;
    switch (dslashFloatPrecision_) {
        case QCU_HALF_PRECISION:
            gauge = fp16Gauge_;
            break;
        case QCU_SINGLE_PRECISION:
            gauge = fp32Gauge_;
            break;
        case QCU_DOUBLE_PRECISION:
            gauge = fp64Gauge_;
            break;
        default:
            errorQcu("Unsupported float precision\n");
    }

    bool default_dagger_flag = false;
    mass_ = mass;
    kappa_ = (1.0 / (2.0 * (4.0 + mass)));

    dslashParam_ = new DslashParam 
                    (
                        default_dagger_flag, dslashFloatPrecision_, nColors_, mInput_,
                        QCU_PARITY::EVEN_PARITY, kappa_, fermionIn_MRHS_, fermionOut_MRHS_,
                        gauge, &lattDesc_, &procDesc_
                    );

    switch (dslashType) {
        case DSLASH_TYPE::DSLASH_WILSON:
            dslash_ = new WilsonDslash(dslashParam_);
            break;

        default:
            errorQcu("Unsupported dslash type\n");
            break;
    }
}

void Qcu::startDslash(int parity, bool daggerFlag) {
    if (nullptr == dslash_) {
        errorQcu("Dslash is not initialized\n");
    }
    if (fermionIn_queue_.size() != mInput_ || fermionOut_queue_.size() != mInput_) {
        errorQcu("Fermion queue is not full\n");
    }
    int Lx = lattDesc_.dims[X_DIM];
    int Ly = lattDesc_.dims[Y_DIM];
    int Lz = lattDesc_.dims[Z_DIM];
    int Lt = lattDesc_.dims[T_DIM];
    dslashParam_->parity = parity;
    dslashParam_->daggerFlag = daggerFlag;

    dslashParam_->fermionIn_MRHS = fermionIn_MRHS_;
    dslashParam_->fermionOut_MRHS = fermionOut_MRHS_;
    // DEBUG

    // lookup table
    void* d_lookup_table_in;
    void* d_lookup_table_out;
    CHECK_CUDA(hipMalloc(&d_lookup_table_in, sizeof(void*) * fermionIn_queue_.size()));    // TODO : fermionIn_queue_.size()改为mInput
    CHECK_CUDA(hipMalloc(&d_lookup_table_out, sizeof(void*) * fermionOut_queue_.size()));

    CHECK_CUDA(hipMemcpy(d_lookup_table_in, fermionIn_queue_.data(), sizeof(void*) * fermionIn_queue_.size(),
                          hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_lookup_table_out, fermionOut_queue_.data(), sizeof(void*) * fermionIn_queue_.size(),
                          hipMemcpyHostToDevice));

    // colorSpinorGather(fermionIn_MRHS_, dslashFloatPrecision_, d_lookup_table_in, inputFloatPrecision_, Lx, Ly, Lz, Lt,
                    //   nColors_, mInput_, NULL);
    TIMER_EVENT(colorSpinorGather(fermionIn_MRHS_, dslashFloatPrecision_, d_lookup_table_in, inputFloatPrecision_, Lx, Ly, Lz, Lt,
                      nColors_, mInput_, NULL), 0, "gather");
    CHECK_CUDA(hipDeviceSynchronize());

    // real op
    int mv_flops = (8 * nColors_ - 2) * nColors_; // (8 * in.Ncolor() - 2) * in.Ncolor();
    int num_mv = Ns / 2;
    double num_op = Lx * Ly * Lz * Lt / 2 * mInput_ * (
        2 * Nd * Ns * nColors_ + 
        2 * Nd * num_mv * mv_flops +
        (2 * Nd - 1) * Ns * nColors_
    );

    double real_num_op = 0; 
    {
        using namespace device;
        int wmma_m = 8;
        int wmma_n = 8;
        int wmma_k = 4;
        int warp_line = div_ceil(nColors_, wmma_m);
        int warp_col = div_ceil(mInput_, wmma_n);
        
        int gemm_flops = wmma_m * wmma_n * (8 * wmma_k - 2);

        real_num_op = Lx * Ly * Lz * Lt / 2 * 8 * warp_line * warp_col *(
            // combination
            double(2 * wmma_m * wmma_k * 2) + // 2个矩阵
            // gemm
            double(2 * gemm_flops) +        // 2个gemm
            // add
            double(4 * wmma_m * wmma_n * 2)  // 4个add
        );
        // printf("warp_line = %d, warp_col = %d, gemm_flops = %d, op = %lf\n", warp_line, warp_col, gemm_flops, real_num_op);
        // printf("part1 : %lf, part2 : %lf, part3 : %lf\n", double(2 * wmma_m * wmma_k * 2), double(2 * gemm_flops), double(4 * wmma_m * wmma_n * 2));
    }
    
    TIMER_EVENT(dslash_->apply(), num_op, "wilson dslash");
    // TIMER_EVENT(dslash_->apply(), real_num_op, "wilson dslash real op");

    TIMER_EVENT(colorSpinorScatter(d_lookup_table_out, inputFloatPrecision_, fermionOut_MRHS_, dslashFloatPrecision_, Lx, Ly, Lz,
                       Lt, nColors_, mInput_, NULL), 0, "scatter");
    // colorSpinorScatter(d_lookup_table_out, inputFloatPrecision_, fermionOut_MRHS_, dslashFloatPrecision_, Lx, Ly, Lz,
    //                    Lt, nColors_, mInput_, NULL);
    CHECK_CUDA(hipDeviceSynchronize());

    CHECK_CUDA(hipFree(d_lookup_table_in));
    CHECK_CUDA(hipFree(d_lookup_table_out));
    fermionIn_queue_.clear();
    fermionOut_queue_.clear();
}

void Qcu::loadGauge(void* gauge, QCU_PRECISION floatPrecision) {
    gauge_ = gauge;
    int Lx = lattDesc_.dims[X_DIM];
    int Ly = lattDesc_.dims[Y_DIM];
    int Lz = lattDesc_.dims[Z_DIM];
    int Lt = lattDesc_.dims[T_DIM];
    int complex_vector_length = Nd * Lx * Ly * Lz * Lt * nColors_ * nColors_;
    
    assert(floatPrecision == QCU_DOUBLE_PRECISION || floatPrecision == QCU_SINGLE_PRECISION ||
           floatPrecision == QCU_HALF_PRECISION);
    // fp64Gauge_ = gauge;
    // CHECK_CUDA(hipMemcpy(fp64Gauge_, gauge, sizeof(double) * 2 * complex_vector_length, hipMemcpyDeviceToDevice));
    copyComplexVector_interface(fp64Gauge_, QCU_DOUBLE_PRECISION, gauge_, floatPrecision, complex_vector_length);
    copyComplexVector_interface(fp32Gauge_, QCU_SINGLE_PRECISION, gauge_, floatPrecision, complex_vector_length);
    copyComplexVector_interface(fp16Gauge_, QCU_HALF_PRECISION, gauge_, floatPrecision, complex_vector_length);
}

void Qcu::pushBackFermions(void* fermionOut, void* fermionIn) {
    if (fermionIn_queue_.size() >= mInput_ || fermionOut_queue_.size() >= mInput_) {
        errorQcu("Fermion queue is full\n");
    }
    fermionIn_queue_.push_back(fermionIn);
    fermionOut_queue_.push_back(fermionOut);
}
}  // namespace qcu
