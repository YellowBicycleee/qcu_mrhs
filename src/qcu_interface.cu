#include <hip/hip_fp16.h>

#include <cassert>
#include <cstdlib>

#include "../tests/public_complex_vector.h"
#include "data_format/qcu_data_format_shift.cuh"
#include "qcu_interface.h"
#include "qcu_public.h"
#include "qcu_utils.h"          // div_ceil
#include "qcu_wmma_constant.h"  // use this to debug
#include "solver/bicgstab.cuh"
#include "timer/timer.h"

#include "check_error/check_cuda.cuh"

#include "lqcd_read_write.h"
#include "precondition/even_odd_precondition.h"
namespace qcu {

void Qcu::allocateMemory() {
    int Lx = lattDesc_.dims[X_DIM];
    int Ly = lattDesc_.dims[Y_DIM];
    int Lz = lattDesc_.dims[Z_DIM];
    int Lt = lattDesc_.dims[T_DIM];

    int vol = Lx * Ly * Lz * Lt;
    int colorSpinorMrhs_size = vol * Ns * nColors_ * mInput_;  // even and odd
    int gauge_size = Nd * vol * nColors_ * nColors_;   // even and odd

    switch (iterateFloatPrecision_) {
        case QCU_HALF_PRECISION: {
            CHECK_CUDA(hipMalloc(&fermionIn_MRHS_, 2 * colorSpinorMrhs_size * sizeof(half)));
            CHECK_CUDA(hipMalloc(&fermionOut_MRHS_, 2 * colorSpinorMrhs_size * sizeof(half)));
        } break;
        case QCU_SINGLE_PRECISION: {
            CHECK_CUDA(hipMalloc(&fermionIn_MRHS_, 2 * colorSpinorMrhs_size * sizeof(float)));
            CHECK_CUDA(hipMalloc(&fermionOut_MRHS_, 2 * colorSpinorMrhs_size * sizeof(float)));
        } break;
        case QCU_DOUBLE_PRECISION: {
            CHECK_CUDA(hipMalloc(&fermionIn_MRHS_, 2 * colorSpinorMrhs_size * sizeof(double)));
            CHECK_CUDA(hipMalloc(&fermionOut_MRHS_, 2 * colorSpinorMrhs_size * sizeof(double)));
        } break;

        default:
            break;
    }
    // gauge field
    CHECK_CUDA(hipMalloc(&fp64Gauge_, 2 * gauge_size * sizeof(double)));
    CHECK_CUDA(hipMalloc(&fp32Gauge_, 2 * gauge_size * sizeof(float)));
    CHECK_CUDA(hipMalloc(&fp16Gauge_, 2 * gauge_size * sizeof(half)));

    CHECK_CUDA(hipMalloc(&d_lookup_table_in_, sizeof(void*) * mInput_));
    CHECK_CUDA(hipMalloc(&d_lookup_table_out_, sizeof(void*) * mInput_));
}

void Qcu::freeMemory() {
    if (dslashParam_ != nullptr) {
        delete dslashParam_;
    }
    if (dslash_ != nullptr) {
        delete dslash_;
    }

    if (fp64Gauge_ != nullptr) {
        CHECK_CUDA(hipFree(fp64Gauge_));
    }
    if (fp32Gauge_ != nullptr) {
        CHECK_CUDA(hipFree(fp32Gauge_));
    }
    if (fp16Gauge_ != nullptr) {
        CHECK_CUDA(hipFree(fp16Gauge_));
    }
    if (fermionIn_MRHS_ != nullptr) {
        CHECK_CUDA(hipFree(fermionIn_MRHS_));
    }
    if (fermionOut_MRHS_ != nullptr) {
        CHECK_CUDA(hipFree(fermionOut_MRHS_));
    }

    if (d_lookup_table_in_ != nullptr) {
        CHECK_CUDA(hipFree(d_lookup_table_in_));
    }

    if (d_lookup_table_out_ != nullptr) {
        CHECK_CUDA(hipFree(d_lookup_table_out_));
    }
}

void Qcu::getDslash(DSLASH_TYPE dslashType, double mass) {
    if (nullptr != dslash_) {
        delete dslash_;
    }
    void* gauge;
    switch (iterateFloatPrecision_) {
        case QCU_HALF_PRECISION:
            gauge = fp16Gauge_;
            break;
        case QCU_SINGLE_PRECISION:
            gauge = fp32Gauge_;
            break;
        case QCU_DOUBLE_PRECISION:
            gauge = fp64Gauge_;
            break;
        default:
            errorQcu("Unsupported float precision\n");
    }

    bool default_dagger_flag = false;
    mass_ = mass;
    kappa_ = (1.0 / (2.0 * (4.0 + mass)));

    dslashParam_ = new DslashParam 
                    (
                        default_dagger_flag, iterateFloatPrecision_, nColors_, mInput_,
                        QCU_PARITY::EVEN_PARITY, kappa_, fermionIn_MRHS_, fermionOut_MRHS_,
                        gauge, &lattDesc_, &procDesc_
                    );

    switch (dslashType) {
        case DSLASH_TYPE::DSLASH_WILSON:
            dslash_ = new WilsonDslash(dslashParam_);
            break;

        default: {
          errorQcu("Unsupported dslash type\n");
          break;
        }

    }
}

void Qcu::startDslash(int parity, bool daggerFlag) {
    if (nullptr == dslash_) {
        errorQcu("Dslash is not initialized\n");
    }
    if (fermionIn_queue_.size() != mInput_ || fermionOut_queue_.size() != mInput_) {
        errorQcu("Fermion queue is not full\n");
    }
    const int Lx = lattDesc_.dims[X_DIM];
    const int Ly = lattDesc_.dims[Y_DIM];
    const int Lz = lattDesc_.dims[Z_DIM];
    const int Lt = lattDesc_.dims[T_DIM];
    dslashParam_->parity = parity;
    dslashParam_->daggerFlag = daggerFlag;

    dslashParam_->fermionIn_MRHS = fermionIn_MRHS_;
    dslashParam_->fermionOut_MRHS = fermionOut_MRHS_;

    CHECK_CUDA(hipMemcpy(d_lookup_table_in_, fermionIn_queue_.data(), sizeof(void*) * fermionIn_queue_.size(),
                          hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_lookup_table_out_, fermionOut_queue_.data(), sizeof(void*) * fermionIn_queue_.size(),
                          hipMemcpyHostToDevice));

    TIMER_EVENT(colorSpinorGather(fermionIn_MRHS_, iterateFloatPrecision_, d_lookup_table_in_,
                                outputFloatPrecision_, Lx, Ly, Lz, Lt,
                      nColors_, mInput_, NULL), 0, "gather");
    CHECK_CUDA(hipDeviceSynchronize());

    // real op
    int mv_flops = (8 * nColors_ - 2) * nColors_; // (8 * in.Ncolor() - 2) * in.Ncolor();
    int num_mv = Ns / 2;
    double num_op = Lx * Ly * Lz * Lt / 2 * mInput_ * (
        2 * Nd * Ns * nColors_ + 
        2 * Nd * num_mv * mv_flops +
        (2 * Nd - 1) * Ns * nColors_
    );

    double real_num_op = 0; 
    {
        using namespace device;
        int wmma_m = 8;
        int wmma_n = 8;
        int wmma_k = 4;
        int warp_line = div_ceil(nColors_, wmma_m);
        int warp_col = div_ceil(mInput_, wmma_n);
        
        int gemm_flops = wmma_m * wmma_n * (8 * wmma_k - 2);

        real_num_op = Lx * Ly * Lz * Lt / 2 * 8 * warp_line * warp_col *(
            // combination
            double(2 * wmma_m * wmma_k * 2) + // 2个矩阵
            // gemm
            double(2 * gemm_flops) +        // 2个gemm
            // add
            double(4 * wmma_m * wmma_n * 2)  // 4个add
        );
    }
    
    TIMER_EVENT(dslash_->apply(), num_op, "wilson dslash");
    TIMER_EVENT(colorSpinorScatter(d_lookup_table_out_, outputFloatPrecision_, fermionOut_MRHS_,
                              iterateFloatPrecision_, Lx, Ly, Lz,
                              Lt, nColors_, mInput_, NULL), 0, "scatter");
    CHECK_CUDA(hipDeviceSynchronize());

    fermionIn_queue_.clear();
    fermionOut_queue_.clear();
}

void Qcu::loadGauge(void* gauge, QCU_PRECISION floatPrecision) {
    gauge_ = gauge;
    int Lx = lattDesc_.dims[X_DIM];
    int Ly = lattDesc_.dims[Y_DIM];
    int Lz = lattDesc_.dims[Z_DIM];
    int Lt = lattDesc_.dims[T_DIM];
    int complex_vector_length = Nd * Lx * Ly * Lz * Lt * nColors_ * nColors_;
    
    assert(floatPrecision == QCU_DOUBLE_PRECISION || floatPrecision == QCU_SINGLE_PRECISION ||
           floatPrecision == QCU_HALF_PRECISION);
    copyComplexVector_interface(fp64Gauge_, QCU_DOUBLE_PRECISION, gauge_, floatPrecision, complex_vector_length);
    copyComplexVector_interface(fp32Gauge_, QCU_SINGLE_PRECISION, gauge_, floatPrecision, complex_vector_length);
    copyComplexVector_interface(fp16Gauge_, QCU_HALF_PRECISION, gauge_, floatPrecision, complex_vector_length);
}

void Qcu::pushBackFermions(void* fermionOut, void* fermionIn) {
    if (fermionIn_queue_.size() >= mInput_ || fermionOut_queue_.size() >= mInput_) {
        errorQcu("Fermion queue is full\n");
    }
    fermionIn_queue_.push_back(fermionIn);
    fermionOut_queue_.push_back(fermionOut);
}



void Qcu::solveFermions(int max_iteration, double max_precision) {
  const int Lx = lattDesc_.dims[X_DIM];
  const int Ly = lattDesc_.dims[Y_DIM];
  const int Lz = lattDesc_.dims[Z_DIM];
  const int Lt = lattDesc_.dims[T_DIM];
  const int vol = Lx * Ly * Lz * Lt;
  const int colorSpinor_len = Ns * nColors_;

  if (mInput_ != fermionIn_queue_.size()) {
    errorQcu("number of fermion is different from mInput\n");
  } else {
    printf("numbers matched, now begin bicg\n");
  }

//   void* d_lookup_table_in;
//   void* d_lookup_table_out;
//   CHECK_CUDA(hipMalloc(&d_lookup_table_in, sizeof(void*) * mInput_));
//   CHECK_CUDA(hipMalloc(&d_lookup_table_out, sizeof(void*) * mInput_));

  vector<void*> fermionIn_queue_odd(fermionIn_queue_.size());
  vector<void*> fermionOut_queue_odd(fermionOut_queue_.size());
  for (int i = 0; i < fermionIn_queue_.size(); i++) {
    fermionIn_queue_odd[i] = static_cast<Complex<OutputFloat>*>(fermionIn_queue_[i]) + colorSpinor_len * vol / 2;
    fermionOut_queue_odd[i] = static_cast<Complex<OutputFloat>*>(fermionOut_queue_[i]) + colorSpinor_len * vol / 2;
  }

  void* fermionIn_MRHS_even = fermionIn_MRHS_;
  void* fermionIn_MRHS_odd = static_cast<Complex<OutputFloat>*>(fermionIn_MRHS_)
                                + colorSpinor_len * mInput_ * vol / 2;
  // gather even
  CHECK_CUDA(hipMemcpy(d_lookup_table_in_,  fermionIn_queue_.data(), sizeof(void*) * mInput_, hipMemcpyHostToDevice));
  TIMER_EVENT(colorSpinorGather(fermionIn_MRHS_even, iterateFloatPrecision_,
                                d_lookup_table_in_,   outputFloatPrecision_,
                                Lx, Ly, Lz, Lt, nColors_, mInput_, NULL)
      , 0, "gather");

  // gather odd
  CHECK_CUDA(hipMemcpy(d_lookup_table_in_, fermionIn_queue_odd.data(), sizeof(void*) * mInput_, hipMemcpyHostToDevice));
  TIMER_EVENT(colorSpinorGather(fermionIn_MRHS_odd, iterateFloatPrecision_,
                                d_lookup_table_in_, outputFloatPrecision_,
                                Lx, Ly, Lz, Lt, nColors_, mInput_, NULL)
      , 0, "gather");



  // SOLVE
  void* gauge;
  if (outputFloatPrecision_ == QCU_DOUBLE_PRECISION) {
    gauge = fp64Gauge_;
  }
  else if (outputFloatPrecision_ == QCU_SINGLE_PRECISION) {
    gauge = fp32Gauge_;
  }
  else {
    gauge = fp16Gauge_;
  }
  qcu::solver::BiCGStabParam param{
    .nColor         = nColors_,
    .mInput         = mInput_,
    .kappa          = kappa_,
    .output_x_mrhs  = fermionOut_MRHS_,
    .input_b_mrhs   = fermionIn_MRHS_,
    .gauge          = gauge,
    .lattDesc       = &lattDesc_,
    .procDesc       = &procDesc_,
    .stream1        = nullptr,
    .stream2        = nullptr
  };
  solver::ApplyBicgStab(param, outputFloatPrecision_, iterateFloatPrecision_,
                            max_iteration, max_precision);

  // scatter
  void* fermionOut_MRHS_even = fermionOut_MRHS_;
  void* fermionOut_MRHS_odd = static_cast<Complex<OutputFloat>*>(fermionOut_MRHS_)
                                  + colorSpinor_len * mInput_ * vol / 2;
  // scatter even
  CHECK_CUDA(hipMemcpy(d_lookup_table_out_, fermionOut_queue_.data(), sizeof(void*) * mInput_, hipMemcpyHostToDevice));
  TIMER_EVENT(
    colorSpinorScatter( d_lookup_table_out_,   outputFloatPrecision_,
                        fermionOut_MRHS_even, iterateFloatPrecision_,
                        Lx, Ly, Lz, Lt, nColors_, mInput_, NULL),
    0, "scatter");
  // scatter odd
  CHECK_CUDA(hipMemcpy(d_lookup_table_out_, fermionOut_queue_odd.data(), sizeof(void*) * mInput_, hipMemcpyHostToDevice));
  TIMER_EVENT(
    colorSpinorScatter( d_lookup_table_out_,  outputFloatPrecision_,
                        fermionOut_MRHS_odd, iterateFloatPrecision_,
                        Lx, Ly, Lz, Lt, nColors_, mInput_, NULL),
    0, "scatter");
  CHECK_CUDA(hipStreamSynchronize(NULL));
  // free lookup-table
//   CHECK_CUDA(hipFree(d_lookup_table_in));
//   CHECK_CUDA(hipFree(d_lookup_table_out));
  fermionIn_queue_.clear();
  fermionOut_queue_.clear();
}

void Qcu::readGaugeFromFile (const char* file_path, void* data_ptr) {
    std::string file = file_path;
    QcuHeader qcuHeader;
    MPI_Desc mpi_desc;
    Latt_Desc latt_desc;

#pragma unroll 
    for (int i = 0; i < Nd; ++i) {
        mpi_desc.data[i] = 1;   // 
        latt_desc.data[i] = lattDesc_.dims[i];
    }
    // MPI_Coordinate: todo 
    MPI_Coordinate coord;
#pragma unroll
    for (int i = 0; i < Nd; ++i) {
        coord.data[i] = 0;
    }

    GaugeReader gaugeReader(file, qcuHeader, mpi_desc, coord);
    qcuHeader.m_lattice_desc.detail();
#pragma unroll
    for (int i = 0; i < Nd; ++i) {
        assert(lattDesc_.dims[i] == qcuHeader.m_lattice_desc.data[i]);
    }
//     qcuHeader.m_lattice_desc.detail();

    auto gauge_length = qcuHeader.GaugeLength();
    std::cout << gauge_length << std::endl;
    Complex<double>* host_ptr = new Complex<double>[gauge_length];
    gaugeReader.read_gauge(reinterpret_cast<std::complex<double>*>(host_ptr), 0);
    Complex<double>* unpreconditioned;
    CHECK_CUDA (hipMalloc(&unpreconditioned, sizeof(Complex<double>) * gauge_length));
    CHECK_CUDA(hipMemcpy(unpreconditioned, host_ptr, sizeof(Complex<double>) * gauge_length, hipMemcpyHostToDevice));
    qcu::GaugeEOPreconditioner<double> preconditioner;
    preconditioner.reverse(static_cast<Complex<double>*>(data_ptr), 
                            unpreconditioned, 
                            latt_desc, 
                            qcuHeader.GaugeSiteLength(),
                            4,  
                            nullptr);

    CHECK_CUDA(hipFree(unpreconditioned));
    delete[] host_ptr;
}

}  // namespace qcu
