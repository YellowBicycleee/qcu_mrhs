#include "hip/hip_runtime.h"
#include <iostream> 
#include <complex>
#include <hipblas.h>
#include "timer.h"
#include <vector>
#include <numeric>
#include "qcu_blas/qcu_transpose_2d.h"
using namespace std;

// 比较两种形式的内积计算耗时
// 1. 交叉点积
// 2. 转置 + 点积 + 转置

#define CHECK_CUBLAS(cmd)   \
do {  \
  hipblasStatus_t stat = cmd;      \
  check_cublas (stat, __FILE__, __LINE__);\
} while (0)

inline void check_cublas (hipblasStatus_t stat, const char* file, int line) {
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf("IN file %s, line %d, error happened\n", file, line);
    abort();
  }
}

complex<double> cpu_inner_dot_c (complex<double>* a, complex<double>* b, int N, int stride) {
  complex<double> res = 0;
  for (int i = 0; i < N; ++i) {
    res += (conj(a[i * stride]) * b[i * stride]);
  }
  return res;
}

void init_complex_1 (complex<double>* a, int N) {
  for (int i = 0; i < N; ++i ) {
    // a[i] = complex<double> (2 * i % 32, 2 * i % 32 + 1);
    a[i] = complex<double> (rand()%16, rand()%16 + 1);
  }
}
void init_complex_2 (complex<double>* a, int N) {
  for (int i = 0; i < N; ++i ) {
    // a[i] = complex<double> (2 * i % 16, 2 * i % 16 + 1);
    a[i] = complex<double> (rand()%16, rand()%16 + 1);
  }
}

bool check_correct (const vector<complex<double>>& a, const vector<complex<double>>& b) {
  int size = a.size();
  if (a.size() != b.size()) {
    return false;
  }
  // bool res = true;
  for (int i = 0; i < size; ++i) {
    if (fabs(a[i].real() - b[i].real()) > 1e-12 || 
        fabs(a[i].imag() - b[i].imag()) > 1e-12 
    ) {
      return false;
    }
  }
  return true;
}

int main () {
  complex<double>* h_a;
  complex<double>* h_b;
  complex<double> h_res;
  complex<double> d_res;

  complex<double>* d_a;
  complex<double>* d_b;
  complex<double>* transposed_d_a;
  complex<double>* transposed_d_b;
  complex<double>* dd_res;

  // malloc 
  const int single_vec_length = 1024 * 1024;
  const int num_vecs = 12;
  const int vector_length = single_vec_length * num_vecs;

  vector<complex<double>> d_res_vec (num_vecs);
  vector<complex<double>> h_res_vec (num_vecs);


  h_a = new complex<double>[vector_length];
  h_b = new complex<double>[vector_length];

  hipMalloc ((void**)&d_a, sizeof(complex<double>) * vector_length);
  hipMalloc ((void**)&d_b, sizeof(complex<double>) * vector_length);
  hipMalloc ((void**)&transposed_d_a, sizeof(complex<double>) * vector_length);
  hipMalloc ((void**)&transposed_d_b, sizeof(complex<double>) * vector_length);
  hipMalloc ((void**)&dd_res, sizeof(complex<double>));

  // init arr
  init_complex_1(h_a, vector_length);
  init_complex_2(h_b, vector_length);
  // copy
  hipMemcpy(d_a, h_a, sizeof(complex<double>) * vector_length, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, sizeof(complex<double>) * vector_length, hipMemcpyHostToDevice);




  // gpu
  // hipblasStatus_t cublas_stat;
  hipblasHandle_t cublas_handle;

  CHECK_CUBLAS (hipblasCreate(&cublas_handle));

    // warm up
  CHECK_CUBLAS (hipblasZdotc(cublas_handle, vector_length, 
                        reinterpret_cast<const hipDoubleComplex*>(d_a), 1, 
                        reinterpret_cast<const hipDoubleComplex*>(d_b), 1, 
                        reinterpret_cast<hipDoubleComplex*>(dd_res)
                  ));

  // =========================================

  // 交叉点积
  d_res_vec.clear();
  h_res_vec.clear();
  cout << "=========stride inner pro, stride = num_vecs =============" << endl;
  // gpu stride innerproduct
  TIMER_EVENT (
    for (int i = 0; i < num_vecs; ++i) {
      CHECK_CUBLAS (hipblasZdotc(cublas_handle, single_vec_length, 
                      reinterpret_cast<const hipDoubleComplex*>(d_a + i), num_vecs, 
                      reinterpret_cast<const hipDoubleComplex*>(d_b + i), num_vecs, 
                      reinterpret_cast<hipDoubleComplex*>(dd_res)
                )
      );
      hipMemcpy(&d_res, dd_res, sizeof(complex<double>), hipMemcpyDeviceToHost);
      d_res_vec.push_back(d_res);
    }
    , 
    "cubulas, stride innerproduct    : "
  );

  // stride innerproduct
  TIMER_EVENT (
    for (int i = 0; i < num_vecs; ++i) {
        h_res = cpu_inner_dot_c(h_a + i, h_b + i, single_vec_length, num_vecs);
        h_res_vec.push_back(h_res);
    }
    ,
    "cpu, stride innerproduct        : "
  );
  cout << "if_res_correct? : " << check_correct(d_res_vec, h_res_vec) << endl;
  cout << "stride = " << num_vecs << ", res = " << std::accumulate(d_res_vec.begin(), d_res_vec.end(), complex<double>(0, 0)) << endl;
  cout << "stride = " << num_vecs << ", res = " << std::accumulate(h_res_vec.begin(), h_res_vec.end(), complex<double>(0, 0)) << endl;



  // 转置 + 点积 + 转置

  // stride = 1 res
  cout << "=========stride inner pro, stride = num_vecs =============" << endl;
  d_res_vec.clear();
  h_res_vec.clear();
  // gpu stride innerproduct
  Timer timer;
  timer.start();

  qcu::qcu_blas::Transpose2D<double2>::Transpose2DParam param {
    single_vec_length, 
    num_vecs, 
    static_cast<void*>(transposed_d_b), 
    static_cast<void*>(d_b)
  };
  hipDeviceSynchronize();
      
  qcu::qcu_blas::Transpose2D<double2> trans2d (param);
  trans2d ();

  param.input = static_cast<void*>(d_a);
  param.output = static_cast<void*>(transposed_d_a);
  trans2d ();
  hipDeviceSynchronize();
  TIMER_EVENT (
    for (int i = 0; i < num_vecs; ++i) 
    {
      CHECK_CUBLAS (hipblasZdotc(cublas_handle, single_vec_length, 
                      reinterpret_cast<const hipDoubleComplex*>(transposed_d_a + i * single_vec_length), 1, 
                      reinterpret_cast<const hipDoubleComplex*>(transposed_d_b + i * single_vec_length), 1, 
                      // reinterpret_cast<const hipDoubleComplex*>(d_a + i * single_vec_length), 1, 
                      // reinterpret_cast<const hipDoubleComplex*>(d_b + i * single_vec_length), 1,
                      reinterpret_cast<hipDoubleComplex*>(dd_res)
                )
      );
      // hipDeviceSynchronize();
      hipMemcpy(&d_res, dd_res, sizeof(complex<double>), hipMemcpyDeviceToHost);
      d_res_vec.push_back(d_res);
    }
    , 
    "gpu, stride = 1 innerproduct    : "
  );

  timer.stop();
  cout << "gpu, stride innerproduct    : " << timer.getElapsedTimeSecond() << "s" << endl;

  // cpu 
    // stride innerproduct
  TIMER_EVENT (
    for (int i = 0; i < num_vecs; ++i) {
        h_res = cpu_inner_dot_c(h_a + i * single_vec_length, h_b + i * single_vec_length, single_vec_length, 1);
        // h_res_vec.push_back(h_res);
        h_res_vec.push_back(h_res);
        // cout << h_res_vec.size() << endl;
    }
    , 
    "cpu, stride innerproduct        : "
  );

  cout << "if_res_correct? : " << check_correct(d_res_vec, h_res_vec) << endl;
  cout << "stride = 1, res = " << std::accumulate(d_res_vec.begin(), d_res_vec.end(), complex<double>(0, 0)) << endl;
  cout << "stride = 1, res = " << std::accumulate(h_res_vec.begin(), h_res_vec.end(), complex<double>(0, 0)) << endl;

  cout << check_correct(d_res_vec, vector<complex<double>>{});
  CHECK_CUBLAS (hipblasDestroy(cublas_handle));
  hipFree(d_a);
  hipFree(d_b);
  hipFree(dd_res);
  delete[] h_a;
  delete[] h_b;
}
