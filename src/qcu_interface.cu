#include <hip/hip_fp16.h>

#include <cassert>
#include <cstdlib>

#include "../tests/public_complex_vector.h"
#include "data_format/qcu_data_format_shift.cuh"
#include "qcu_interface.h"
#include "qcu_public.h"
#include "qcu_utils.h"          // div_ceil
#include "qcu_wmma_constant.h"  // use this to debug
#include "solver/bicgstab.cuh"
#include "timer/timer.h"

#include "check_error/check_cuda.cuh"

#include "lqcd_read_write.h"
#include "precondition/even_odd_precondition.h"

#include "qcu_blas/qcu_blas.h"

namespace qcu {

void Qcu::allocateMemory() {
    int Lx = lattice_desc_.data[X_DIM];
    int Ly = lattice_desc_.data[Y_DIM];
    int Lz = lattice_desc_.data[Z_DIM];
    int Lt = lattice_desc_.data[T_DIM];

    int vol = Lx * Ly * Lz * Lt;
    int colorSpinorMrhs_size = vol * Ns * n_colors_ * m_input_;  // even and odd
    int gauge_size = Nd * vol * n_colors_ * n_colors_;   // even and odd

    switch (compute_floatprecision_) {
        case QCU_HALF_PRECISION: {
            CHECK_CUDA(hipMalloc(&fermion_in_mrhs_, 2 * colorSpinorMrhs_size * sizeof(half)));
            CHECK_CUDA(hipMalloc(&fermion_out_mrhs_, 2 * colorSpinorMrhs_size * sizeof(half)));
        } break;
        case QCU_SINGLE_PRECISION: {
            CHECK_CUDA(hipMalloc(&fermion_in_mrhs_, 2 * colorSpinorMrhs_size * sizeof(float)));
            CHECK_CUDA(hipMalloc(&fermion_out_mrhs_, 2 * colorSpinorMrhs_size * sizeof(float)));
        } break;
        case QCU_DOUBLE_PRECISION: {
            CHECK_CUDA(hipMalloc(&fermion_in_mrhs_, 2 * colorSpinorMrhs_size * sizeof(double)));
            CHECK_CUDA(hipMalloc(&fermion_out_mrhs_, 2 * colorSpinorMrhs_size * sizeof(double)));
        } break;

        default:
            break;
    }
    // gauge field
    CHECK_CUDA(hipMalloc(&fp64_gauge_, 2 * gauge_size * sizeof(double)));
    CHECK_CUDA(hipMalloc(&fp32_gauge_, 2 * gauge_size * sizeof(float)));
    CHECK_CUDA(hipMalloc(&fp16_gauge_, 2 * gauge_size * sizeof(half)));

    CHECK_CUDA(hipMalloc(&d_lookup_table_in_, sizeof(void*) * m_input_));
    CHECK_CUDA(hipMalloc(&d_lookup_table_out_, sizeof(void*) * m_input_));
}

void Qcu::freeMemory() {
    if (dslash_param_ != nullptr) {
        delete dslash_param_;
    }
    if (dslash_ != nullptr) {
        delete dslash_;
    }

    if (fp64_gauge_ != nullptr) {
        CHECK_CUDA(hipFree(fp64_gauge_));
    }
    if (fp32_gauge_ != nullptr) {
        CHECK_CUDA(hipFree(fp32_gauge_));
    }
    if (fp16_gauge_ != nullptr) {
        CHECK_CUDA(hipFree(fp16_gauge_));
    }
    if (fermion_in_mrhs_ != nullptr) {
        CHECK_CUDA(hipFree(fermion_in_mrhs_));
    }
    if (fermion_out_mrhs_ != nullptr) {
        CHECK_CUDA(hipFree(fermion_out_mrhs_));
    }

    if (d_lookup_table_in_ != nullptr) {
        CHECK_CUDA(hipFree(d_lookup_table_in_));
    }

    if (d_lookup_table_out_ != nullptr) {
        CHECK_CUDA(hipFree(d_lookup_table_out_));
    }
}

void Qcu::get_dslash(DSLASH_TYPE dslashType, double mass) {
    if (nullptr != dslash_) {
        delete dslash_;
    }
    void* gauge;
    switch (compute_floatprecision_) {
        case QCU_HALF_PRECISION:
            gauge = fp16_gauge_;
            break;
        case QCU_SINGLE_PRECISION:
            gauge = fp32_gauge_;
            break;
        case QCU_DOUBLE_PRECISION:
            gauge = fp64_gauge_;
            break;
        default:
            errorQcu("Unsupported float precision\n");
    }

    bool default_dagger_flag = false;
    mass_ = mass;
    kappa_ = (1.0 / (2.0 * (4.0 + mass)));

    dslash_param_ = new DslashParam
                    (
                        default_dagger_flag, compute_floatprecision_, n_colors_, m_input_,
                        QCU_PARITY::EVEN_PARITY, kappa_, fermion_in_mrhs_, fermion_out_mrhs_,
                        gauge, &lattice_desc_, &process_desc_
                    );

    switch (dslashType) {
        case DSLASH_TYPE::DSLASH_WILSON:
            dslash_ = new WilsonDslash(dslash_param_);
            break;

        default: {
          errorQcu("Unsupported dslash type\n");
          break;
        }

    }
}

void Qcu::start_dslash(int parity, bool daggerFlag) {
    if (nullptr == dslash_) {
        errorQcu("Dslash is not initialized\n");
    }
    if (fermion_in_queue_.size() != m_input_ || fermion_out_vec_.size() != m_input_) {
        errorQcu("Fermion queue is not full\n");
    }
// #define DEBUG_INTERFACE
#ifdef DEBUG_INTERFACE
    std::cout << "DEBUG_INFO: fermionInQueue:";
    std::cout << "fermion.size() = " << fermion_in_queue_.size() << std::endl;
    for (int i = 0; i < fermion_in_queue_.size(); ++i) {
        std::cout << fermion_in_queue_[i] << " ";
    }
    std::cout << std::endl;
    std::cout << "DEBUG_INFO: fermionOutQueue:";
    for (int i = 0; i < fermion_out_vec_.size(); ++i) {
        std::cout << fermion_out_vec_[i] << " ";
    }
    std::cout << std::endl;
#endif
    const int Lx = lattice_desc_.data[X_DIM];
    const int Ly = lattice_desc_.data[Y_DIM];
    const int Lz = lattice_desc_.data[Z_DIM];
    const int Lt = lattice_desc_.data[T_DIM];
    dslash_param_->parity = parity;
    dslash_param_->daggerFlag = daggerFlag;

    dslash_param_->fermionIn_MRHS = fermion_in_mrhs_;
    dslash_param_->fermionOut_MRHS = fermion_out_mrhs_;

    CHECK_CUDA(hipMemcpy(d_lookup_table_in_, fermion_in_queue_.data(), sizeof(void*) * fermion_in_queue_.size(),
                          hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_lookup_table_out_, fermion_out_vec_.data(), sizeof(void*) * fermion_in_queue_.size(),
                          hipMemcpyHostToDevice));

    TIMER_EVENT(colorSpinorGather(fermion_in_mrhs_, compute_floatprecision_, d_lookup_table_in_,
                                out_float_precision_, Lx, Ly, Lz, Lt,
                      n_colors_, m_input_, NULL), 0, "gather");
    CHECK_CUDA(hipDeviceSynchronize());

    // real op
    int mv_flops = (8 * n_colors_ - 2) * n_colors_; // (8 * in.Ncolor() - 2) * in.Ncolor();
    int num_mv = Ns / 2;
    double num_op = Lx * Ly * Lz * Lt / 2 * m_input_ * (
        2 * Nd * Ns * n_colors_ +
        2 * Nd * num_mv * mv_flops +
        (2 * Nd - 1) * Ns * n_colors_
    );

    [[maybe_unused]] double real_num_op = 0;
    {
        using namespace device;
        int wmma_m = 8;
        int wmma_n = 8;
        int wmma_k = 4;
        int warp_line = div_ceil(n_colors_, wmma_m);
        int warp_col = div_ceil(m_input_, wmma_n);
        
        int gemm_flops = wmma_m * wmma_n * (8 * wmma_k - 2);

        real_num_op = Lx * Ly * Lz * Lt / 2 * 8 * warp_line * warp_col *(
            // combination
            double(2 * wmma_m * wmma_k * 2) + // 2个矩阵
            // gemm
            double(2 * gemm_flops) +        // 2个gemm
            // add
            double(4 * wmma_m * wmma_n * 2)  // 4个add
        );
    }
    
    TIMER_EVENT(dslash_->apply(), num_op, "wilson dslash");
    TIMER_EVENT(colorSpinorScatter(d_lookup_table_out_, out_float_precision_, fermion_out_mrhs_,
                              compute_floatprecision_, Lx, Ly, Lz,
                              Lt, n_colors_, m_input_, NULL), 0, "scatter");
    CHECK_CUDA(hipDeviceSynchronize());

    fermion_in_queue_.clear();
    fermion_out_vec_.clear();
}

void Qcu::mat_qcu (bool daggerFlag) {
    if (nullptr == dslash_) {
        errorQcu("Dslash is not initialized\n");
    }
    if (fermion_in_queue_.size() != m_input_ || fermion_out_vec_.size() != m_input_) {
        errorQcu("Fermion queue is not full\n");
    }

    const int Lx = lattice_desc_.data[X_DIM];
    const int Ly = lattice_desc_.data[Y_DIM];
    const int Lz = lattice_desc_.data[Z_DIM];
    const int Lt = lattice_desc_.data[T_DIM];
    // dslash_param_->parity = parity;
    dslash_param_->daggerFlag = daggerFlag;
    dslash_param_->fermionIn_MRHS = fermion_in_mrhs_;
    dslash_param_->fermionOut_MRHS = fermion_out_mrhs_;

    Complex host_kappa = Complex<OutputFloat>(kappa_, 0);
    CHECK_CUDA(hipMalloc(&device_kappa_, sizeof(Complex<OutputFloat>) ));
    CHECK_CUDA(hipMemcpy(device_kappa_, &host_kappa, sizeof(Complex<OutputFloat>), hipMemcpyHostToDevice));

    vector<void*> fermion_in_half (fermion_in_queue_.size());
    vector<void*> fermion_out_half (fermion_out_vec_.size());
    const int vol = Lx * Ly * Lz * Lt;
    const int fermion_half_len = (vol / 2) * Ns * n_colors_ * m_input_;
    // mat_qcu = fermionIn - kappa fermionOut   
    qcu::qcu_blas::Complex_xsay<OutputFloat> xsay_op;

    for (int parity =0; parity < 2; ++parity) {
        dslash_param_->parity = parity;
        for (int i = 0; i < m_input_; ++i) {
            fermion_out_half[i] = static_cast<Complex<OutputFloat>*>(fermion_out_vec_[i]) + parity * fermion_half_len;
            fermion_in_half[i] = static_cast<Complex<OutputFloat>*>(fermion_in_queue_[i]) + (1 - parity) * fermion_half_len;
        }
        CHECK_CUDA(
            hipMemcpy(d_lookup_table_in_, fermion_in_half.data(), sizeof(void*) * m_input_, hipMemcpyHostToDevice)
        );
        CHECK_CUDA(
            hipMemcpy(d_lookup_table_out_, fermion_out_half.data(), sizeof(void*) * m_input_, hipMemcpyHostToDevice)
        );
        colorSpinorGather(fermion_in_mrhs_, compute_floatprecision_, d_lookup_table_in_, out_float_precision_,
            Lx, Ly, Lz, Lt, n_colors_, m_input_, NULL);
        CHECK_CUDA(hipDeviceSynchronize());

        dslash_->apply();
        
        CHECK_CUDA(hipDeviceSynchronize());

        // qcu::qcu_blas::Complex_xsay<OutputFloat>::Complex_xsayArgument arg (
        //     static_cast<Complex<OutputFloat>*>(fermion_out_mrhs_),   // Complex<_Float>* res,
        //     static_cast<Complex<OutputFloat>*>(fermion_in_mrhs_),    // Complex<_Float>* x,
        //     static_cast<Complex<OutputFloat>*>(device_kappa_),      // Complex<_Float>* a,
        //     static_cast<Complex<OutputFloat>*>(fermion_out_mrhs_),   // Complex<_Float>* y,
        //     fermion_half_len,                                       // int single_vec_len,
        //     1,                                                      // int inc_idx,
        //     nullptr                                                 // hipStream_t stream = nullptr
        // );
        // xsay_op(arg);

        colorSpinorScatter(d_lookup_table_out_, out_float_precision_, fermion_out_mrhs_, compute_floatprecision_,
            Lx, Ly, Lz, Lt, n_colors_, m_input_, NULL);
        CHECK_CUDA(hipDeviceSynchronize());
    }

    CHECK_CUDA(
        hipMemcpy(d_lookup_table_in_, fermion_in_queue_.data(), sizeof(void*) * m_input_, hipMemcpyHostToDevice)
    );
    CHECK_CUDA(
        hipMemcpy(d_lookup_table_out_, fermion_out_vec_.data(), sizeof(void*) * m_input_, hipMemcpyHostToDevice)
    );
    colorSpinorGather(fermion_in_mrhs_, compute_floatprecision_, d_lookup_table_in_, out_float_precision_,
            Lx * 2, Ly, Lz, Lt, n_colors_, m_input_, NULL);
    colorSpinorGather(fermion_out_mrhs_, compute_floatprecision_, d_lookup_table_out_, out_float_precision_,
            Lx * 2, Ly, Lz, Lt, n_colors_, m_input_, NULL);
    qcu::qcu_blas::Complex_xsay<OutputFloat>::Complex_xsayArgument arg (
        static_cast<Complex<OutputFloat>*>(fermion_out_mrhs_),   // Complex<_Float>* res,
        static_cast<Complex<OutputFloat>*>(fermion_in_mrhs_),    // Complex<_Float>* x,
        static_cast<Complex<OutputFloat>*>(device_kappa_),      // Complex<_Float>* a,
        static_cast<Complex<OutputFloat>*>(fermion_out_mrhs_),   // Complex<_Float>* y,
        fermion_half_len * 2,                                       // int single_vec_len,
        1,                                                      // int inc_idx,
        nullptr                                                 // hipStream_t stream = nullptr
    );
    xsay_op(arg);
    colorSpinorScatter(d_lookup_table_out_, out_float_precision_, fermion_out_mrhs_, compute_floatprecision_,
            Lx * 2, Ly, Lz, Lt, n_colors_, m_input_, NULL);
    
    CHECK_CUDA(hipFree(device_kappa_));
    fermion_in_queue_.clear();
    fermion_out_vec_.clear();
}
void Qcu::load_gauge(void* gauge, QCU_PRECISION floatPrecision) {
    gauge_external_ = gauge;
    int Lx = lattice_desc_.data[X_DIM];
    int Ly = lattice_desc_.data[Y_DIM];
    int Lz = lattice_desc_.data[Z_DIM];
    int Lt = lattice_desc_.data[T_DIM];
    int complex_vector_length = Nd * Lx * Ly * Lz * Lt * n_colors_ * n_colors_;
    
    assert(floatPrecision == QCU_DOUBLE_PRECISION || floatPrecision == QCU_SINGLE_PRECISION ||
           floatPrecision == QCU_HALF_PRECISION);
    copyComplexVector_interface(fp64_gauge_, QCU_DOUBLE_PRECISION, gauge_external_, floatPrecision, complex_vector_length);
    copyComplexVector_interface(fp32_gauge_, QCU_SINGLE_PRECISION, gauge_external_, floatPrecision, complex_vector_length);
    copyComplexVector_interface(fp16_gauge_, QCU_HALF_PRECISION, gauge_external_, floatPrecision, complex_vector_length);
}

void Qcu::push_back_fermion(void* fermionOut, void* fermionIn) {
    if (fermion_in_queue_.size() >= m_input_ || fermion_out_vec_.size() >= m_input_) {
        errorQcu("Fermion queue is full\n");
    }
    fermion_in_queue_.push_back(fermionIn);
    fermion_out_vec_.push_back(fermionOut);
}



void Qcu::solve_fermions(int max_iteration, double max_precision) {
  const int Lx = lattice_desc_.data[X_DIM];
  const int Ly = lattice_desc_.data[Y_DIM];
  const int Lz = lattice_desc_.data[Z_DIM];
  const int Lt = lattice_desc_.data[T_DIM];
  const int vol = Lx * Ly * Lz * Lt;
  const int colorSpinor_len = Ns * n_colors_;

  if (m_input_ != fermion_in_queue_.size()) {
    errorQcu("number of fermion is different from mInput\n");
  } else {
    printf("numbers matched, now begin bicg\n");
  }

//   void* d_lookup_table_in;
//   void* d_lookup_table_out;
//   CHECK_CUDA(hipMalloc(&d_lookup_table_in, sizeof(void*) * m_input_));
//   CHECK_CUDA(hipMalloc(&d_lookup_table_out, sizeof(void*) * m_input_));

  vector<void*> fermionIn_queue_odd(fermion_in_queue_.size());
  vector<void*> fermionOut_queue_odd(fermion_out_vec_.size());
  for (int i = 0; i < fermion_in_queue_.size(); i++) {
    fermionIn_queue_odd[i] = static_cast<Complex<OutputFloat>*>(fermion_in_queue_[i]) + colorSpinor_len * vol / 2;
    fermionOut_queue_odd[i] = static_cast<Complex<OutputFloat>*>(fermion_out_vec_[i]) + colorSpinor_len * vol / 2;
  }

  void* fermionIn_MRHS_even = fermion_in_mrhs_;
  void* fermionIn_MRHS_odd = static_cast<Complex<OutputFloat>*>(fermion_in_mrhs_)
                                + colorSpinor_len * m_input_ * vol / 2;
  // gather even
  CHECK_CUDA(hipMemcpy(d_lookup_table_in_,  fermion_in_queue_.data(), sizeof(void*) * m_input_, hipMemcpyHostToDevice));
  TIMER_EVENT(colorSpinorGather(fermionIn_MRHS_even, compute_floatprecision_,
                                d_lookup_table_in_,   out_float_precision_,
                                Lx, Ly, Lz, Lt, n_colors_, m_input_, NULL)
      , 0, "gather");

  // gather odd
  CHECK_CUDA(hipMemcpy(d_lookup_table_in_, fermionIn_queue_odd.data(), sizeof(void*) * m_input_, hipMemcpyHostToDevice));
  TIMER_EVENT(colorSpinorGather(fermionIn_MRHS_odd, compute_floatprecision_,
                                d_lookup_table_in_, out_float_precision_,
                                Lx, Ly, Lz, Lt, n_colors_, m_input_, NULL)
      , 0, "gather");



  // SOLVE
  void* gauge;
  if (out_float_precision_ == QCU_DOUBLE_PRECISION) {
    gauge = fp64_gauge_;
  }
  else if (out_float_precision_ == QCU_SINGLE_PRECISION) {
    gauge = fp32_gauge_;
  }
  else {
    gauge = fp16_gauge_;
  }
  qcu::solver::BiCGStabParam param{
    .nColor         = n_colors_,
    .mInput         = m_input_,
    .kappa          = kappa_,
    .output_x_mrhs  = fermion_out_mrhs_,
    .input_b_mrhs   = fermion_in_mrhs_,
    .gauge          = gauge,
    .lattDesc       = &lattice_desc_,
    .procDesc       = &process_desc_,
    .stream1        = nullptr,
    .stream2        = nullptr
  };
  solver::ApplyBicgStab(param, out_float_precision_, compute_floatprecision_,
                            max_iteration, max_precision);

  // scatter
  void* fermionOut_MRHS_even = fermion_out_mrhs_;
  void* fermionOut_MRHS_odd = static_cast<Complex<OutputFloat>*>(fermion_out_mrhs_)
                                  + colorSpinor_len * m_input_ * vol / 2;
  // scatter even
  CHECK_CUDA(hipMemcpy(d_lookup_table_out_, fermion_out_vec_.data(), sizeof(void*) * m_input_, hipMemcpyHostToDevice));
  TIMER_EVENT(
    colorSpinorScatter( d_lookup_table_out_,   out_float_precision_,
                        fermionOut_MRHS_even, compute_floatprecision_,
                        Lx, Ly, Lz, Lt, n_colors_, m_input_, NULL),
    0, "scatter");
  // scatter odd
  CHECK_CUDA(hipMemcpy(d_lookup_table_out_, fermionOut_queue_odd.data(), sizeof(void*) * m_input_, hipMemcpyHostToDevice));
  TIMER_EVENT(
    colorSpinorScatter( d_lookup_table_out_,  out_float_precision_,
                        fermionOut_MRHS_odd, compute_floatprecision_,
                        Lx, Ly, Lz, Lt, n_colors_, m_input_, NULL),
    0, "scatter");
  CHECK_CUDA(hipStreamSynchronize(NULL));
  // free lookup-table
//   CHECK_CUDA(hipFree(d_lookup_table_in));
//   CHECK_CUDA(hipFree(d_lookup_table_out));
  fermion_in_queue_.clear();
  fermion_out_vec_.clear();
}

void Qcu::read_gauge_from_file (const char* file_path, void* data_ptr) {
    std::string file = file_path;
    QcuHeader qcuHeader;
    MPI_Desc mpi_desc;
    Latt_Desc latt_desc;

#pragma unroll 
    for (int i = 0; i < Nd; ++i) {
        mpi_desc.data[i] = 1;   // 
        latt_desc.data[i] = lattice_desc_.data[i];
    }
    // MPI_Coordinate: todo 
    MPI_Coordinate coord;
#pragma unroll
    for (int i = 0; i < Nd; ++i) {
        coord.data[i] = 0;
    }

    GaugeReader gaugeReader(file, qcuHeader, mpi_desc, coord, latt_desc);

    auto gauge_length = qcuHeader.GaugeLength();
    // std::cout << gauge_length << std::endl; 
    Complex<double>* host_ptr = new Complex<double>[gauge_length];
    gaugeReader.read_gauge(reinterpret_cast<std::complex<double>*>(host_ptr), 0);
    Complex<double>* unpreconditioned;
    CHECK_CUDA (hipMalloc(&unpreconditioned, sizeof(Complex<double>) * gauge_length));
    CHECK_CUDA(hipMemcpy(unpreconditioned, host_ptr, sizeof(Complex<double>) * gauge_length, hipMemcpyHostToDevice));
    qcu::GaugeEOPreconditioner<double> preconditioner;
    preconditioner.reverse(static_cast<Complex<double>*>(data_ptr), 
                            unpreconditioned, 
                            latt_desc, 
                            qcuHeader.GaugeSiteLength(),
                            4,  
                            nullptr);

    CHECK_CUDA(hipFree(unpreconditioned));
    delete[] host_ptr;
}

}  // namespace qcu
