#include "hip/hip_runtime.h"
#include "kernel/transpose_2dim.cuh"
#include "qcu_blas/qcu_transpose_2d.h"
#include "qcu_utils.h"
#include "qcu_macro.h"

#include "complex/qcu_complex.cuh"
#include <hip/hip_fp16.h>
#include <iostream>
// #include <cuda_device_runtime_api.h>
namespace qcu::qcu_blas {
template 
void Transpose2D<double2>::operator () ();

template
void Transpose2D<float2>::operator () ();

template
void Transpose2D<half2>::operator () ();

template
void Transpose2D<double>::operator () ();

template
void Transpose2D<float>::operator () ();

template
void Transpose2D<int>::operator () ();

template <typename _Tp>
void Transpose2D<_Tp>::operator () () {
  int device_id;
  hipDeviceProp_t prop;
  CHECK_CUDA(hipGetDevice(&device_id));
  CHECK_CUDA(hipGetDeviceProperties(&prop, device_id));
  int max_block_x = prop.maxGridSize[0];// .maxThreadsPerBlock;
  int max_block_y = prop.maxGridSize[0];// .maxThreadsPerBlock;

  // std::cout << "max_threads_per_block = " << prop.maxGridSize[0] << std::endl;
  // std::cout << "max_threads_per_block = " << prop.maxGridSize[1] << std::endl;
  // std::cout << "max_threads_per_block = " << prop.maxGridSize[2] << std::endl;



  int threads_per_block_x = qcu::device::kernel::N_TILE;
  int threads_per_block_y = qcu::device::kernel::M_TILE;
  int blocks_per_grid_y = std::min(div_ceil(param.m, threads_per_block_y), max_block_y);
  int blocks_per_grid_x = std::min(div_ceil(param.n, threads_per_block_x), max_block_x / max_block_y);

  dim3 block_size(threads_per_block_x, threads_per_block_y);
  dim3 grid_size(blocks_per_grid_x, blocks_per_grid_y);
  
  // std::cout << "block_size = " << block_size.x << " " << block_size.y << std::endl;
  // std::cout << "grid_size = " << grid_size.x << " " << grid_size.y << std::endl;
  // std::cout << "param.m = " << param.m << std::endl;
  // std::cout << "param.n = " << param.n << std::endl;
  device::kernel::transpose2D_kernel<_Tp> <<< grid_size, block_size >>>
    (static_cast<_Tp*> (param.output), static_cast<_Tp*> (param.input), param.m, param.n);
  CHECK_CUDA(hipGetLastError());

}


}