//
// Created by Wang Jiancheng on 24-10-21.
//
// Create this file to record variables used frequently in the Qcu class,
//        to avoid repeated computation and improve efficiency.

#include <cstdint>

#include "desc/qcu_desc.h"
#include "qcu_config/qcu_config.h"
#include "qcu_helper.h"
#include "check_error/check_cuda.cuh"
namespace qcu::config {

constexpr int kQcuCudaStreamNum = 9;

static hipStream_t stream_pack[kQcuCudaStreamNum] = {nullptr};
static qcu::QcuLattDesc lattice_desc; // record the lattice size in single process (rather than total lattice)
static qcu::QcuProcDesc process_desc;

int32_t lattice_volume() {
    return lattice_desc.lattice_volume();
}
int32_t whole_lattice_volume() {
    return lattice_desc.lattice_volume() * process_desc.process_volume();
}
bool set_config(int Lx, int Ly, int Lz, int Lt, int Gx, int Gy, int Gz, int Gt){
    lattice_desc = qcu::QcuLattDesc(Lx, Ly, Lz, Lt);
    process_desc = qcu::QcuProcDesc(Gx, Gy, Gz, Gt);
    return true;
}

qcu::QcuLattDesc* get_lattice_desc_ptr() {
    return &lattice_desc;
}
qcu::QcuProcDesc* get_process_desc_ptr() {
    return &process_desc;
}

// cuda stream functions
constexpr int get_qcu_stream_num() noexcept {
    return kQcuCudaStreamNum;
}
constexpr hipStream_t* get_qcu_stream_ptr() noexcept {
    return stream_pack;
}
hipStream_t get_qcu_default_stream() noexcept {
    return stream_pack[0];
}
void init_streams() {
#pragma unroll
    for (int i = 0; i < kQcuCudaStreamNum; ++i) {
        CHECK_CUDA(hipStreamCreate(&stream_pack[i]));
    }
}
void destroy_streams() {
#pragma unroll
    for (int i = 0; i < kQcuCudaStreamNum; ++i) {
        CHECK_CUDA(hipStreamDestroy(stream_pack[i]));
    }
}

}
