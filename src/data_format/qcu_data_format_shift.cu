#include <cstdio>

#include "data_format/qcu_data_format_shift.cuh"
#include "kernel/copy_vector/copy_color_spinor.cuh"
#include "kernel/copy_vector/copy_complex_vector.cuh"
#include "qcu_enum.h"
namespace qcu {

template <typename DestFloat, typename SrcFloat>
static void copyVector_Complex(void* __restrict__ dst, void* __restrict__ src, int complex_vector_length,
                        hipStream_t stream) {
    using DestFloat2 = typename qcu::Float2Wrapper<DestFloat>::Float2;
    using SrcFloat2 = typename qcu::Float2Wrapper<SrcFloat>::Float2;
    int block_size = 256;
    int grid_size = complex_vector_length / block_size;
    device::copyComplexVector<DestFloat2, SrcFloat2>
        <<<grid_size, block_size>>>(static_cast<DestFloat2*>(dst), static_cast<SrcFloat2*>(src), complex_vector_length);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipStreamSynchronize(stream));
}

template <typename DestFloat, typename SrcFloat>
static void copyVector_Complex_Async(void* __restrict__ dst, void* __restrict__ src, int complex_vector_length,
                              hipStream_t stream) {
    using DestFloat2 = typename qcu::Float2Wrapper<DestFloat>::Float2;
    using SrcFloat2 = typename qcu::Float2Wrapper<SrcFloat>::Float2;
    int block_size = 256;
    int grid_size = complex_vector_length / block_size;
    device::copyComplexVector<DestFloat2, SrcFloat2>
        <<<grid_size, block_size>>>(static_cast<DestFloat2*>(dst), static_cast<SrcFloat2*>(src), complex_vector_length);
    CHECK_CUDA(hipGetLastError());
}

template <typename DstFloat, typename SrcFloat>
void colorSpinorScatter(void* __restrict__ global_dst_array, void* __restrict__ global_src_ptr, int Lx, int Ly, int Lz,
                        int Lt, int n_color, int m_input, hipStream_t stream) {
    using DstFloat2 = typename qcu::Float2Wrapper<DstFloat>::Float2;
    using SrcFloat2 = typename qcu::Float2Wrapper<SrcFloat>::Float2;
    int block_size = 256;
    int grid_size = (Lx / 2 * Ly * Lz * Lt + block_size - 1) / block_size;
    device::color_spinor_scatter_kernel<DstFloat2, SrcFloat2>
        <<<grid_size, block_size>>>(static_cast<DstFloat2**>(global_dst_array), static_cast<SrcFloat2*>(global_src_ptr),
                                    Lx, Ly, Lz, Lt, n_color, m_input);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipStreamSynchronize(stream));
}

template <typename DstFloat, typename SrcFloat>
void colorSpinorGather(void* __restrict__ global_dst_ptr, void* __restrict__ global_src_array, int Lx, int Ly, int Lz,
                       int Lt, int n_color, int m_input, hipStream_t stream) {
    using DstFloat2 = typename qcu::Float2Wrapper<DstFloat>::Float2;
    using SrcFloat2 = typename qcu::Float2Wrapper<SrcFloat>::Float2;
    int block_size = 256;
    int grid_size = (Lx * Ly * Lz * Lt / 2 + block_size - 1) / block_size;
    printf("DEBUG file %s, line %d, global_src_array = %p, global_dst_ptr = %p\n", __FILE__, __LINE__,global_src_array, global_dst_ptr);
    device::color_spinor_gather_kernel<DstFloat2, SrcFloat2>
        <<<grid_size, block_size>>>(static_cast<DstFloat2*>(global_dst_ptr), static_cast<SrcFloat2**>(global_src_array),
                                    Lx, Ly, Lz, Lt, n_color, m_input);

    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipStreamSynchronize(stream));
}

template <typename DstFloat>
static void instantiate_copyVector_Complex_SrcFloat(void* __restrict__ dest, void* __restrict__ src,
                                                    QCU_PRECISION srcPrec, int complex_vector_length,
                                                    hipStream_t stream) {
    switch (srcPrec) {
        case QCU_HALF_PRECISION:
            copyVector_Complex<DstFloat, half>(dest, src, complex_vector_length, stream);
            break;
        case QCU_SINGLE_PRECISION:
            copyVector_Complex<DstFloat, float>(dest, src, complex_vector_length, stream);
            break;
        case QCU_DOUBLE_PRECISION:
            copyVector_Complex<DstFloat, double>(dest, src, complex_vector_length, stream);
            break;
        default:
            errorQcu("Unsupported Source Float precision\n");
    };
}

template <typename DstFloat>
static void instantiate_colorSpinorScatter_SrcFloat(void* __restrict__ global_dst_array,
                                                    void* __restrict__ global_src_ptr, QCU_PRECISION srcPrec, int Lx,
                                                    int Ly, int Lz, int Lt, int n_color, int m_input,
                                                    hipStream_t stream) {
    switch (srcPrec) {
        case QCU_HALF_PRECISION:
            colorSpinorScatter<DstFloat, half>(global_dst_array, global_src_ptr, Lx, Ly, Lz, Lt, n_color, m_input,
                                               stream);
            break;
        case QCU_SINGLE_PRECISION:
            colorSpinorScatter<DstFloat, float>(global_dst_array, global_src_ptr, Lx, Ly, Lz, Lt, n_color, m_input,
                                                stream);
            break;
        case QCU_DOUBLE_PRECISION:
            colorSpinorScatter<DstFloat, double>(global_dst_array, global_src_ptr, Lx, Ly, Lz, Lt, n_color, m_input,
                                                 stream);
            break;
        default:
            errorQcu("Unsupported Source Float precision\n");
    };
}

template <typename DstFloat>
static void instantiate_colorSpinorGather_SrcFloat(void* __restrict__ global_dst_ptr,
                                                   void* __restrict__ global_src_array, QCU_PRECISION srcPrec, int Lx,
                                                   int Ly, int Lz, int Lt, int n_color, int m_input,
                                                   hipStream_t stream) {
    switch (srcPrec) {
        case QCU_HALF_PRECISION:
            colorSpinorGather<DstFloat, half>(global_dst_ptr, global_src_array, Lx, Ly, Lz, Lt, n_color, m_input,
                                              stream);
            break;
        case QCU_SINGLE_PRECISION:
            colorSpinorGather<DstFloat, float>(global_dst_ptr, global_src_array, Lx, Ly, Lz, Lt, n_color, m_input,
                                               stream);
            break;
        case QCU_DOUBLE_PRECISION:
            colorSpinorGather<DstFloat, double>(global_dst_ptr, global_src_array, Lx, Ly, Lz, Lt, n_color, m_input,
                                                stream);
            break;
        default:
            errorQcu("Unsupported Destination Float precision\n");
    }
}

void copyComplexVector_interface(void* __restrict__ dest, QCU_PRECISION destPrec, void* __restrict__ src,
                                 QCU_PRECISION srcPrec, int complex_vector_length, hipStream_t stream) {
    if (destPrec == QCU_PRECISION_UNDEFINED || srcPrec == QCU_PRECISION_UNDEFINED) {
        errorQcu("Undefined precision\n");
    }
    // instantiate the template function
    switch (destPrec) {
        case QCU_HALF_PRECISION:
            instantiate_copyVector_Complex_SrcFloat<half>(dest, src, srcPrec, complex_vector_length, stream);
            break;
        case QCU_SINGLE_PRECISION:
            instantiate_copyVector_Complex_SrcFloat<float>(dest, src, srcPrec, complex_vector_length, stream);
            break;
        case QCU_DOUBLE_PRECISION:
            instantiate_copyVector_Complex_SrcFloat<double>(dest, src, srcPrec, complex_vector_length, stream);
            break;
        default:
            errorQcu("Unsupported Destination Float precision\n");
            break;
    }
}

void colorSpinorScatter(void* __restrict__ global_dst_array, QCU_PRECISION dstPrec, void* __restrict__ global_src_ptr,
                        QCU_PRECISION srcPrec, int Lx, int Ly, int Lz, int Lt, int n_color, int m_input,
                        hipStream_t stream) {
    switch (dstPrec) {
        case QCU_HALF_PRECISION:
            instantiate_colorSpinorScatter_SrcFloat<half>(global_dst_array, global_src_ptr, srcPrec, Lx, Ly, Lz, Lt,
                                                          n_color, m_input, stream);
            break;
        case QCU_SINGLE_PRECISION:
            instantiate_colorSpinorScatter_SrcFloat<float>(global_dst_array, global_src_ptr, srcPrec, Lx, Ly, Lz, Lt,
                                                           n_color, m_input, stream);
            break;
        case QCU_DOUBLE_PRECISION:
            instantiate_colorSpinorScatter_SrcFloat<double>(global_dst_array, global_src_ptr, srcPrec, Lx, Ly, Lz, Lt,
                                                            n_color, m_input, stream);
            break;
        default:
            errorQcu("Unsupported Destination Float precision\n");
            break;
    }
}

void colorSpinorGather(void* __restrict__ global_dst_ptr, QCU_PRECISION dstPrec, void* __restrict__ global_src_array,
                       QCU_PRECISION srcPrec, int Lx, int Ly, int Lz, int Lt, int n_color, int m_input,
                       hipStream_t stream) {
    switch (dstPrec) {
        case QCU_HALF_PRECISION:
            instantiate_colorSpinorGather_SrcFloat<half>(global_dst_ptr, global_src_array, srcPrec, Lx, Ly, Lz, Lt,
                                                         n_color, m_input, stream);
            break;
        case QCU_SINGLE_PRECISION:
            instantiate_colorSpinorGather_SrcFloat<float>(global_dst_ptr, global_src_array, srcPrec, Lx, Ly, Lz, Lt,
                                                          n_color, m_input, stream);
            break;
        case QCU_DOUBLE_PRECISION:
            instantiate_colorSpinorGather_SrcFloat<double>(global_dst_ptr, global_src_array, srcPrec, Lx, Ly, Lz, Lt,
                                                           n_color, m_input, stream);
            break;
        default:
            errorQcu("Unsupported Destination Float precision\n");
            break;
    }
}
}  // namespace qcu
