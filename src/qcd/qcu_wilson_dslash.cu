#include <hip/hip_fp16.h>

#include "kernel/su_n_m_rhs_dslash.cuh"
#include "qcd/qcu_dslash.h"
#include "qcu_public.h"
#include "qcu_wmma_constant.h"
#include "check_error/check_cuda.cuh"
namespace qcu {

// clang-format off
template <typename Float>
inline void ApplyWilsonDslash_Mrhs( Float* __restrict__ out, Float* __restrict__ in, Float* __restrict__ gauge, 
                                    int Lx, int Ly, int Lz, int Lt, int g_x, int g_y, int g_z, int g_t, 
                                    int parity, bool dagger_flag, int n_color, int m_rhs, hipStream_t& stream
) {
    // clang-format on
    int vol = Lx * Ly * Lz * Lt / 2;
    int warp_num_per_block = WARP_PER_BLOCK;

    dim3 block_size(WARP_SIZE, warp_num_per_block);
    dim3 grid_size(vol);
    device::wilson_dslash_su_n_mrhs<Float> <<<grid_size, block_size, 0, stream>>>(
        out, in, gauge, Lx, Ly, Lz, Lt, g_x, g_y, g_z, g_t, parity, dagger_flag, n_color, m_rhs);
}

void WilsonDslash::apply() {
    int Lx = dslashParam_->lattDesc->data[X_DIM];
    int Ly = dslashParam_->lattDesc->data[Y_DIM];
    int Lz = dslashParam_->lattDesc->data[Z_DIM];
    int Lt = dslashParam_->lattDesc->data[T_DIM];

    int g_x = dslashParam_->procDesc->data[X_DIM];
    int g_y = dslashParam_->procDesc->data[Y_DIM];
    int g_z = dslashParam_->procDesc->data[Z_DIM];
    int g_t = dslashParam_->procDesc->data[T_DIM];

    // clang-format off
    switch (dslashParam_->precision) {
        case QCU_HALF_PRECISION:
            ApplyWilsonDslash_Mrhs<half>(static_cast<half*>(dslashParam_->fermionOut_MRHS), 
                                         static_cast<half*>(dslashParam_->fermionIn_MRHS), 
                                         static_cast<half*>(dslashParam_->gauge), 
                                         Lx, Ly, Lz, Lt, g_x, g_y, g_z, g_t, 
                                         dslashParam_->parity, dslashParam_->daggerFlag, 
                                         dslashParam_->nColor, dslashParam_->mInput, 
                                         dslashParam_->stream1);
            /* code */
            break;
        case QCU_SINGLE_PRECISION:
            errorQcu("Not implemented yet\n");  // TODO
            assert(0);
            break;
        case QCU_DOUBLE_PRECISION:
            // printf("Double precision, FILE %s LINE %d FERMION_OUT_MRHS = %p, FERMION_IN_MRHS = %p\n",
                                            //  __FILE__, __LINE__, dslashParam_->fermionOut_MRHS, dslashParam_->fermionIn_MRHS);
            ApplyWilsonDslash_Mrhs<double>(static_cast<double*>(dslashParam_->fermionOut_MRHS), 
                                           static_cast<double*>(dslashParam_->fermionIn_MRHS), 
                                           static_cast<double*>(dslashParam_->gauge), 
                                           Lx, Ly, Lz, Lt, g_x, g_y, g_z, g_t, 
                                           dslashParam_->parity, dslashParam_->daggerFlag, 
                                           dslashParam_->nColor, dslashParam_->mInput, 
                                           dslashParam_->stream1); 
            break;
        default:
            errorQcu("Not implemented yet\n");  // TODO
            assert(0);
            break;
    }
    CHECK_CUDA(hipStreamSynchronize(dslashParam_->stream1));
    // clang-format on
}
void WilsonDslash::preApply() {
    errorQcu("Not implemented yet\n");  // TODO
}
void WilsonDslash::postApply() {
    errorQcu("Not implemented yet\n");  // TODO
}
// TODO : calc flops
void WilsonDslash::flops() {
    errorQcu("Not implemented yet\n");  // TODO
}

}  // namespace qcu