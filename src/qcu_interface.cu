#include "qcu_interface.cuh"

#include <cassert>
#include <cstdlib>

#include "data_format/qcu_data_format_shift.cuh"
#include "qcu_macro.h"

namespace qcu {

void Qcu::getDslash(DSLASH_TYPE dslashType, double mass) {
    if (nullptr != dslash_) {
        delete dslash_;
    }
    void* gauge;
    switch (dslashFloatPrecision_) {
        case QCU_HALF_PRECISION:
            gauge = fp16Gauge_;
            break;
        case QCU_SINGLE_PRECISION:
            gauge = fp32Gauge_;
            break;
        case QCU_DOUBLE_PRECISION:
            gauge = fp64Gauge_;
            break;
        default:
            errorQcu("Unsupported float precision\n");
    }

    bool default_dagger_flag = false;
    mass_ = mass;
    kappa_ = (1.0 / (2.0 * (4.0 + mass)));

    dslashParam_ = new DslashParam(dslashFloatPrecision_, nColors_, mInput_, kappa_, QCU_PARITY::EVEN_PARITY,
                                   default_dagger_flag, fermionIn_MRHS_, fermionOut_MRHS_, gauge, lattDesc_, procDesc_);

    switch (dslashType) {
        case DSLASH_TYPE::DSLASH_WILSON:
            dslash_ = new WilsonDslash(*dslashParam_);
            break;

        default:
            errorQcu("Unsupported dslash type\n");
            break;
    }
}

void Qcu::startDslash(int parity, bool daggerFlag) {
    if (nullptr == dslash_) {
        errorQcu("Dslash is not initialized\n");
    }
    if (fermionIn_queue_.size() != mInput_ || fermionOut_queue_.size() != mInput_) {
        errorQcu("Fermion queue is not full\n");
    }
    int Lx = lattDesc_.dims[X_DIM];
    int Ly = lattDesc_.dims[Y_DIM];
    int Lz = lattDesc_.dims[Z_DIM];
    int Lt = lattDesc_.dims[T_DIM];
    dslashParam_->parity = parity;
    dslashParam_->daggerFlag = daggerFlag;

    colorSpinorGather(fermionIn_MRHS_, dslashFloatPrecision_, fermionIn_queue_.data(), inputFloatPrecision_, Lx, Ly, Lz,
                      Lt, nColors_, mInput_, NULL);
    dslash_->apply();
    CHECK_CUDA(hipDeviceSynchronize());

    colorSpinorScatter(fermionOut_queue_.data(), inputFloatPrecision_, fermionOut_MRHS_, dslashFloatPrecision_, Lx, Ly,
                       Lz, Lt, nColors_, mInput_, NULL);

    fermionIn_queue_.clear();
    fermionOut_queue_.clear();
}

void Qcu::loadGauge(void* gauge, QCU_PRECISION floatPrecision) {
    gauge_ = gauge;
    int Lx = lattDesc_.dims[X_DIM];
    int Ly = lattDesc_.dims[Y_DIM];
    int Lz = lattDesc_.dims[Z_DIM];
    int Lt = lattDesc_.dims[T_DIM];
    int complex_vector_length = DIRECTIONS * Lx * Ly * Lz * Lt * nColors_ * nColors_;
    assert(floatPrecision == QCU_DOUBLE_PRECISION || floatPrecision == QCU_SINGLE_PRECISION ||
           floatPrecision == QCU_HALF_PRECISION);
    copyComplexVector_interface(fp64Gauge_, QCU_DOUBLE_PRECISION, gauge_, floatPrecision, complex_vector_length);
    copyComplexVector_interface(fp32Gauge_, QCU_SINGLE_PRECISION, gauge_, floatPrecision, complex_vector_length);
    copyComplexVector_interface(fp16Gauge_, QCU_HALF_PRECISION, gauge_, floatPrecision, complex_vector_length);
}

void Qcu::pushBackFermions(void* fermionOut, void* fermionIn) {
    if (fermionIn_queue_.size() >= mInput_ || fermionOut_queue_.size() >= mInput_) {
        errorQcu("Fermion queue is full\n");
    }
    fermionIn_queue_.push_back(fermionIn);
    fermionOut_queue_.push_back(fermionOut);
}
}  // namespace qcu